#include "hip/hip_runtime.h"
#include "mma.h"

#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
//#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>
#include <cmath>
//#include <cstdio>
#include <iomanip>

#define CEIL_DIV(x, k) (((x) + (k)-1) / (k))

float get_sec() {
  struct timeval time;
  gettimeofday(&time, NULL);
  return (1e6 * time.tv_sec + time.tv_usec);
}

float cpu_elapsed_time(float &beg, float &end) { return 1.0e-6 * (end - beg); }

void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};

void CudaDeviceInfo() {
  int deviceId;

  hipGetDevice(&deviceId);

  hipDeviceProp_t props{};
  hipGetDeviceProperties(&props, deviceId);

  printf("Device ID: %d\n\
    Name: %s\n\
    Compute Capability: %d.%d\n\
    memoryBusWidth: %d\n\
    maxThreadsPerBlock: %d\n\
    maxThreadsPerMultiProcessor: %d\n\
    maxRegsPerBlock: %d\n\
    maxRegsPerMultiProcessor: %d\n\
    totalGlobalMem: %zuMB\n\
    sharedMemPerBlock: %zuKB\n\
    sharedMemPerMultiprocessor: %zuKB\n\
    totalConstMem: %zuKB\n\
    multiProcessorCount: %d\n\
    Warp Size: %d\n",
         deviceId, props.name, props.major, props.minor, props.memoryBusWidth,
         props.maxThreadsPerBlock, props.maxThreadsPerMultiProcessor,
         props.regsPerBlock, props.regsPerMultiprocessor,
         props.totalGlobalMem / 1024 / 1024, props.sharedMemPerBlock / 1024,
         props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
         props.multiProcessorCount, props.warpSize);
};

void randomize_matrix(float *mat, int N) {
  // NOTICE: Use gettimeofday instead of srand((unsigned)time(NULL)); the time
  // precision is too low and the same random number is generated.
  struct timeval time {};
  gettimeofday(&time, nullptr);
  srand(time.tv_usec);
  for (int i = 0; i < N; i++) {
    float tmp = (float)(rand() % 5) + 0.01 * (rand() % 5);
    tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
    mat[i] = tmp;
  }
}

bool verify_matrix(float *matRef, float *matOut, int N) {
  double diff = 0.0;
  int i;
  for (i = 0; i < N; i++) {
    diff = std::fabs(matRef[i] - matOut[i]);
    if (diff > 0.01) {
      printf("Divergence! Should %5.2f, Is %5.2f (Diff %5.2f) at %d\n",
             matRef[i], matOut[i], diff, i);
      return false;
    }
  }
  return true;
}

void run_kernel(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
  // Settings for A6000
  const uint K12_NUM_THREADS = 128;
  const uint K12_BN = 128;
  const uint K12_BM = 128;
  const uint K12_BK = 16;
  const uint K12_WN = 64;
  const uint K12_WM = 64;
  const uint K12_WNITER = 4;
  const uint K12_TN = 4;
  const uint K12_TM = 8;
  dim3 blockDim(K12_NUM_THREADS);

  constexpr uint NUM_WARPS = K12_NUM_THREADS / 32;

  // warptile in threadblocktile
  static_assert((K12_BN % K12_WN == 0) and (K12_BM % K12_WM == 0));
  static_assert((K12_BN / K12_WN) * (K12_BM / K12_WM) == NUM_WARPS);

  // threads in warpsubtile
  static_assert((K12_WM * K12_WN) % (WARPSIZE * K12_TM * K12_TN * K12_WNITER) == 0);
  constexpr uint K12_WMITER = (K12_WM * K12_WN) / (32 * K12_TM * K12_TN * K12_WNITER);
  // warpsubtile in warptile
  static_assert((K12_WM % K12_WMITER == 0) and (K12_WN % K12_WNITER == 0));

  static_assert((K12_NUM_THREADS * 4) % K12_BK == 0,
                "NUM_THREADS*4 must be multiple of K9_BK to avoid quantization "
                "issues during GMEM->SMEM tiling (loading only parts of the "
                "final row of Bs during each iteraion)");
  static_assert((K12_NUM_THREADS * 4) % K12_BN == 0,
                "NUM_THREADS*4 must be multiple of K9_BN to avoid quantization "
                "issues during GMEM->SMEM tiling (loading only parts of the "
                "final row of As during each iteration)");
  static_assert(K12_BN % (16 * K12_TN) == 0,
                "BN must be a multiple of 16*TN to avoid quantization effects");
  static_assert(K12_BM % (16 * K12_TM) == 0,
                "BM must be a multiple of 16*TM to avoid quantization effects");
  static_assert((K12_BM * K12_BK) % (4 * K12_NUM_THREADS) == 0,
                "BM*BK must be a multiple of 4*256 to vectorize loads");
  static_assert((K12_BN * K12_BK) % (4 * K12_NUM_THREADS) == 0,
                "BN*BK must be a multiple of 4*256 to vectorize loads");

  dim3 gridDim(CEIL_DIV(N, K12_BN), CEIL_DIV(M, K12_BM));
  runSgemmDoubleBuffering2<K12_BM, K12_BN, K12_BK, K12_WM, K12_WN, K12_WNITER,
                           K12_TM, K12_TN, K12_NUM_THREADS>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

const std::string errLogFile = "matrixValidationFailure.txt";

int main(int argc, char **argv) {
  // get environment variable for device
  int deviceIdx = 0;
  if (getenv("DEVICE") != NULL) {
    deviceIdx = atoi(getenv("DEVICE"));
  }
  cudaCheck(hipSetDevice(deviceIdx));

  // Using cudaEvent for gpu stream timing, cudaEvent is equivalent to
  // publishing event tasks in the target stream
  float elapsed_time;
  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);

  // cuBLAS FLOPs ceiling is reached at 8192
  std::vector<int> SIZE = {128, 256, 512, 1024, 2048, 4096};

  long m, n, k, max_size;
  max_size = SIZE[SIZE.size() - 1];
  std::cout << "Max size: " << max_size << std::endl;

  float alpha = 0.5, beta = 3.0; // GEMM input parameters, C=α*AB+β*C

  float *A = nullptr, *B = nullptr, *C = nullptr; // host matrices
  float *dA = nullptr, *dB = nullptr, *dC = nullptr; // device matrices

  A = (float *)malloc(sizeof(float) * max_size * max_size);
  B = (float *)malloc(sizeof(float) * max_size * max_size);
  C = (float *)malloc(sizeof(float) * max_size * max_size);

  randomize_matrix(A, max_size * max_size);
  randomize_matrix(B, max_size * max_size);
  randomize_matrix(C, max_size * max_size);

  cudaCheck(hipMalloc((void **)&dA, sizeof(float) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dB, sizeof(float) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dC, sizeof(float) * max_size * max_size));

  cudaCheck(hipMemcpy(dA, A, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dB, B, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dC, C, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));

  int repeat_times = 50;
  for (int size : SIZE) {
    m = n = k = size;

    std::cout << "dimensions(m=n=k) " << m << ", alpha: " << alpha
              << ", beta: " << beta << std::endl;
    // Executes the kernel, modifies the result matrix
    run_kernel(m, n, k, alpha, dA, dB, beta, dC);
    cudaCheck(hipDeviceSynchronize());
    cudaCheck(hipGetLastError()); // Check for async errors during kernel run
    hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);

    hipEventRecord(beg);
    for (int j = 0; j < repeat_times; j++) {
      // We don't reset dC between runs to save time
      run_kernel(m, n, k, alpha, dA, dB, beta, dC);
    }
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time /= 1000.; // Convert to seconds

    long flops = 2 * m * n * k;
    printf(
        "Average elapsed time: (%7.6f) s, performance: (%7.1f) GFLOPS. size: "
        "(%ld).\n",
        elapsed_time / repeat_times,
        (repeat_times * flops * 1e-9) / elapsed_time, m);
    fflush(stdout);
    // make dC back to original value.
    cudaCheck(hipMemcpy(dC, C, sizeof(float) * m * n, hipMemcpyHostToDevice));
  }

  // Free up CPU and GPU space
  free(A);
  free(B);
  free(C);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);

  return 0;
};
